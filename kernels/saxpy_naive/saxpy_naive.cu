#include <hip/hip_runtime.h>
#include <stdio.h>

// Naive SAXPY CUDA kernel using vector types
__global__ void saxpy_naive(float4 *x, float4 *y, float a, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i].x = a * x[i].x + y[i].x;
        y[i].y = a * x[i].y + y[i].y;
        y[i].z = a * x[i].z + y[i].z;
        y[i].w = a * x[i].w + y[i].w;
    }
}

int main(int argc, char** argv){
  int n = 1024;
  float a = 2.0f;
  float4* x = new float4[n];
  float4* y = new float4[n];
  for (int i = 0; i < n; i++){
    x[i] = make_float4(i, i+1, i+2, i+3);
  }
  float4* d_x;
  float4* d_y;
  hipMalloc(&d_x, n * sizeof(float4));
  hipMalloc(&d_y, n * sizeof(float4));
  hipMemcpy(d_x, x, n * sizeof(float4), hipMemcpyHostToDevice);
  saxpy_naive<<<1, 1024>>>(d_x, d_y, a, n);
  hipMemcpy(y, d_y, n * sizeof(float4), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++){
    printf("y[%d] = (%f, %f, %f, %f)\n", i, y[i].x, y[i].y, y[i].z, y[i].w);
  }
  hipFree(d_x);
  hipFree(d_y);
  delete[] x;
  return 0;
}